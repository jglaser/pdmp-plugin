#include "hip/hip_runtime.h"
#include "PotentialPairPDMPGPU.cuh"
#include <hoomd/HOOMDMath.h>
#include <hoomd/Index1D.h>
#include <hoomd/ParticleData.cuh>

/*! \file PotentialPairPDMPGPU.cuh
    \brief Defines templated GPU kernel code for calculating the pair forces.
*/

#ifdef NVCC
template<unsigned int compute_virial>
__global__ void gpu_compute_pair_forces_pdmp_kernel(float4 *d_force,
                                               float *d_virial,
                                               const unsigned int virial_pitch,
                                               const unsigned int N,
                                               const Scalar4 *d_pos,
                                               const BoxDim box,
                                               const unsigned int *d_n_neigh,
                                               const unsigned int *d_nlist,
                                               const Index2D nli,
                                               const Scalar *d_params,
                                               const float *d_rcutsq,
                                               const unsigned int ntypes)
    {
    Index2D typpair_idx(ntypes);
    const unsigned int num_typ_parameters = typpair_idx.getNumElements();

    // shared arrays for per type pair parameters
    extern __shared__ char s_data[];
    Scalar *s_params = (Scalar *)(&s_data[0]);
    float *s_rcutsq = (float *)(&s_data[num_typ_parameters*sizeof(Scalar)]);

    // load in the per type pair parameters
    for (unsigned int cur_offset = 0; cur_offset < num_typ_parameters; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < num_typ_parameters)
            {
            s_rcutsq[cur_offset + threadIdx.x] = d_rcutsq[cur_offset + threadIdx.x];
            s_params[cur_offset + threadIdx.x] = d_params[cur_offset + threadIdx.x];
            }
        }
    __syncthreads();

    // start by identifying which particle we are to handle
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    // load in the length of the neighbor list (MEM_TRANSFER: 4 bytes)
    unsigned int n_neigh = d_n_neigh[idx];

    // read in the position of our particle.
    // (MEM TRANSFER: 16 bytes)
    float4 postypei = tex1Dfetch(pdata_pos_tex, idx);
    float3 posi = make_float3(postypei.x, postypei.y, postypei.z);


    // initialize the force to 0
    float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float virialxx = 0.0f;
    float virialxy = 0.0f;
    float virialxz = 0.0f;
    float virialyy = 0.0f;
    float virialyz = 0.0f;
    float virialzz = 0.0f;

    // prefetch neighbor index
    unsigned int cur_j = 0;
    unsigned int next_j = d_nlist[nli(idx, 0)];

    // loop over neighbors
    // on pre Fermi hardware, there is a bug that causes rare and random ULFs when simply looping over n_neigh
    // the workaround (activated via the template paramter) is to loop over nlist.height and put an if (i < n_neigh)
    // inside the loop
    #if (__CUDA_ARCH__ < 200)
    for (int neigh_idx = 0; neigh_idx < nli.getH(); neigh_idx++)
    #else
    for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
    #endif
        {
        #if (__CUDA_ARCH__ < 200)
        if (neigh_idx < n_neigh)
        #endif
            {
            // read the current neighbor index (MEM TRANSFER: 4 bytes)
            // prefetch the next value and set the current one
            cur_j = next_j;
            next_j = d_nlist[nli(idx, neigh_idx+1)];

            // get the neighbor's position (MEM TRANSFER: 16 bytes)
            float4 postypej = tex1Dfetch(pdata_pos_tex, cur_j);
            float3 posj = make_float3(postypej.x, postypej.y, postypej.z);

            // calculate dr (with periodic boundary conditions) (FLOPS: 3)
            float3 dx = posi - posj;

            // apply periodic boundary conditions: (FLOPS 12)
            dx = box.minImage(dx);

            // access the per type pair parameters
            unsigned int typpair = typpair_idx(__float_as_int(postypei.w), __float_as_int(postypej.w));
            float rcutsq = s_rcutsq[typpair];
            Scalar param = s_params[typpair];

            Scalar3 f = make_scalar3(0.0,0.0,0.0);
            // evaluate the potential

            // dimensions of cubic overlap volume
            Scalar Lsq= rcutsq/Scalar(3.0);
            if (dx.x*dx.x < Lsq && dx.y*dx.y < Lsq && dx.z*dx.z < Lsq)
                {
                Scalar L = sqrtf(Lsq);

                Scalar3 pe_factors = make_scalar3(0.0,0.0,0.0);
                pe_factors.x = (Scalar(1.0)-copysignf(Scalar(1.0),dx.x)*dx.x/L);
                pe_factors.y = (Scalar(1.0)-copysignf(Scalar(1.0),dx.y)*dx.y/L);
                pe_factors.z = (Scalar(1.0)-copysignf(Scalar(1.0),dx.z)*dx.z/L);

                Scalar max_energy = param;


                f.x = copysignf(Scalar(1.0),dx.x)/L*max_energy*pe_factors.y*pe_factors.z;
                f.y = copysignf(Scalar(1.0),dx.y)/L*max_energy*pe_factors.x*pe_factors.z;
                f.z = copysignf(Scalar(1.0),dx.z)/L*max_energy*pe_factors.x*pe_factors.y;
                force.x += f.x;
                force.y += f.y;
                force.z += f.z;
                force.w += max_energy*pe_factors.x*pe_factors.y*pe_factors.z;
                }

            // calculate the virial
            if (compute_virial)
                {
                virialxx +=  f.x*dx.x/Scalar(2.0);
                virialxy +=  (f.x*dx.y+f.y*dx.x)/Scalar(4.0);
                virialxz +=  (f.x*dx.z+f.z*dx.x)/Scalar(4.0);
                virialyy +=  f.y*dx.y/Scalar(2.0);
                virialyz +=  (f.y*dx.z+f.z*dx.y)/Scalar(4.0);
                virialzz +=  f.z*dx.z/Scalar(2.0);
                }
            }
        }

    // potential energy per particle must be halved
    force.w *= 0.5f;
    // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
    d_force[idx] = force;

    if (compute_virial)
        {
        d_virial[0*virial_pitch+idx] = virialxx;
        d_virial[1*virial_pitch+idx] = virialxy;
        d_virial[2*virial_pitch+idx] = virialxz;
        d_virial[3*virial_pitch+idx] = virialyy;
        d_virial[4*virial_pitch+idx] = virialyz;
        d_virial[5*virial_pitch+idx] = virialzz;
        }
    }

//! Kernel driver that computes lj forces on the GPU for LJForceComputeGPU
/*! \param pair_args Other arugments to pass onto the kernel
    \param d_params Parameters for the potential, stored per type pair
    
    This is just a driver function for gpu_compute_pair_forces_kernel(), see it for details.
*/
hipError_t gpu_compute_pair_forces_pdmp(float4 *d_force,
              float *d_virial,
              const unsigned int virial_pitch,
              const unsigned int N,
              const unsigned int n_ghost,
              const Scalar4 *d_pos,
              const BoxDim& box,
              const unsigned int *d_n_neigh,
              const unsigned int *d_nlist,
              const Index2D& nli,
              const float *d_rcutsq, 
              const unsigned int ntypes,
              const unsigned int block_size,
              const unsigned int compute_virial,
              const Scalar *d_params)
    {
    assert(d_params);
    
    // setup the grid to run the kernel
    dim3 grid( N / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // bind the position texture
    pdata_pos_tex.normalized = false;
    pdata_pos_tex.filterMode = hipFilterModePoint;
    hipError_t error = hipBindTexture(0, pdata_pos_tex, d_pos, sizeof(Scalar4)*(N+n_ghost));
    if (error != hipSuccess)
        return error;

    Index2D typpair_idx(ntypes);
    unsigned int shared_bytes = (2*sizeof(float) + sizeof(Scalar))
                                * typpair_idx.getNumElements();
    
    // run the kernel
    if (compute_virial)
        gpu_compute_pair_forces_pdmp_kernel<1>
          <<<grid, threads, shared_bytes>>>(d_force, d_virial, virial_pitch, N, d_pos, box, d_n_neigh, d_nlist, nli, d_params, d_rcutsq, ntypes);
    else
        gpu_compute_pair_forces_pdmp_kernel<0>
          <<<grid, threads, shared_bytes>>>(d_force, d_virial, virial_pitch, N, d_pos, box, d_n_neigh, d_nlist, nli, d_params, d_rcutsq, ntypes);
        
    return hipSuccess;
    }
#endif

